#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

/* �{���x����[���{���x�ɕϊ� */
__device__ void double_to_float_float(double a, float *a_hi, float *a_lo)
{
	*a_hi = a;
	*a_lo = a - (float) a;
}

/* �[���{���x����{���x�ɕϊ� */
__device__ void float_float_to_double(float a_hi, float a_lo, double *a)
{
	*a = a_hi + (double) a_lo;
}

/* �[���{���x���Z */
__device__ void float_float_add(float *a_hi, float *a_lo, float b_hi, float b_lo, float c_hi, float c_lo)
{
	float sh, eh, v;

	/* TWO_SUM */
	sh = b_hi + c_hi;
	v = sh - b_hi;
	eh = (b_hi - (sh - v)) + (c_hi - v);

	/* */
	eh += (b_lo + c_lo);

	/* FAST_TWO_SUM */
	*a_hi = sh + eh;
	v = *a_hi - sh;
	*a_lo = (sh - (*a_hi - v)) + (eh - v);

}

/* �[���{���x��Z */
__device__ void float_float_mul(float *a_hi, float *a_lo, float b_hi, float b_lo, float c_hi, float c_lo)
{
	float p1, p2, d_hi, d_lo, e_hi, e_lo, t, v;

	/* TWO_PROD */
	p1 = b_hi * c_hi;

	t = 4097.0 * b_hi;
	d_hi = t - (t - b_hi);
	d_lo = b_hi - d_hi;

	t = 4097.0 * c_hi;
	e_hi = t - (t - c_hi);
	e_lo = c_hi - e_hi;

	p2 = ((d_hi * e_hi - p1) + d_hi * e_lo + d_lo * e_hi) + d_lo * e_lo;

	/* */
	p2 += (b_hi * c_lo) + (b_lo * c_hi);

	/* FAST_TWO_SUM */
	*a_hi = p1 + p2;
	v = *a_hi - p1;
	*a_lo = (p1 - (*a_hi - v)) + (p2 - v);

}

/* �{���x�A�[���{���x��r */
__global__ void kernel()
{
	size_t t_s, t_e;

	double a, a_dash;
	float a_hi, a_lo;
	double theta_a = 2.21315648654123846246;

	double b, b_dash;
	float b_hi, b_lo;
	double theta_b = 1.21315648654123846246;

	double c, c_dash;
	float c_hi, c_lo;
	double theta_c = 0.21315648654123846246;

	/* �N���b�N�֐��Ăяo���R�X�g */
	t_s = clock();
	t_e = clock();
	printf("clock %ld clocks\n\n", t_e - t_s);

	/* �T�C���֐��Ăяo���R�X�g */
	t_s = clock();
	a = sin(theta_a);
	t_e = clock();
	printf("sinf %ld clocks\n\n", t_e - t_s);

	/* �{���x -> �[���{���x�ϊ� */
	t_s = clock();
	double_to_float_float(a, &a_hi, &a_lo);
	t_e = clock();
	printf("double to float-float %ld clocks\n\n", t_e - t_s);

	/* �[���{���x -> �{���x�ϊ� */
	t_s = clock();
	float_float_to_double(a_hi, a_lo, &a_dash);
	t_e = clock();
	printf("float-float to double %ld clocks\n\n", t_e - t_s);

	/* �ϊ����� */
	printf("a  = %1.15e\n", a);
	printf("a' = %1.15e\n\n", a_dash);

	b = sin(theta_b);
	c = sin(theta_c);

	/* ���Z��r */
	printf("Add\n");

	t_s = clock();
	a = b + c;
	t_e = clock();
	printf("double %ld clocks\n", t_e - t_s);

	double_to_float_float(b, &b_hi, &b_lo);
	double_to_float_float(c, &c_hi, &c_lo);

	t_s = clock();
	float_float_add(&a_hi, &a_lo, b_hi, b_lo, c_hi, c_lo);
	t_e = clock();
	printf("float-float %ld clocks\n", t_e - t_s);

	float_float_to_double(a_hi, a_lo, &a_dash);

	printf("a  = %1.15e\n", a);
	printf("a' = %1.15e\n\n", a_dash);


		/* ��Z��r */
		printf("Multiply\n");

	b = -sin(theta_b);
	c = sin(theta_c);

	t_s = clock();
	a = b * c;
	t_e = clock();
	printf("double %ld clocks\n", t_e - t_s);


	double_to_float_float(b, &b_hi, &b_lo);
	double_to_float_float(c, &c_hi, &c_lo);


	t_s = clock();
	float_float_mul(&a_hi, &a_lo, b_hi, b_lo, c_hi, c_lo);
	t_e = clock();
	printf("float-float %ld clocks\n", t_e - t_s);

	float_float_to_double(a_hi, a_lo, &a_dash);


	printf("a  = %1.15e\n", a);
	printf("a' = %1.15e\n\n", a_dash);

	/* ��Βl�v�Z�R�X�g���� */
	long *lp_a_hi, *lp_a_lo;
	lp_a_hi = (long*) &a_hi;
	lp_a_lo = (long*) &a_lo;

	printf("FABS\n");
	t_s = clock();
	*lp_a_hi &= 0x7fffffff;
	*lp_a_lo &= 0x7fffffff;
	t_e = clock();
	printf("float-float %ld clocks\n", t_e - t_s);

	float_float_to_double(a_hi, a_lo, &a_dash);
	printf("a' = %1.15e\n", a_dash);
}

int main()
{
	int device_id = 0; /* ����GPU������ꍇ�ɂ�0�ȊO���ݒ�\ */
	hipSetDevice(device_id);

	kernel << <1, 1 >> >();
	hipDeviceSynchronize();

	return 0;
}